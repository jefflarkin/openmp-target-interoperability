
#include <hip/hip_runtime.h>
__global__
void saxpy_kernel(int n, float a, float *x, float *y)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;

  if ( i < n )
    y[i] += a * x[i];
}
extern "C" void saxpy(int n ,float a, float *x, float *y)
{
  dim3 griddim, blockdim;

  blockdim = dim3(128,1,1);
  griddim = dim3(n/blockdim.x,1,1);

  saxpy_kernel<<<griddim,blockdim>>>(n,a,x,y);
}
