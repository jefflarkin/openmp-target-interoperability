#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <hip/hip_runtime.h>

extern "C" void saxpy(int,float,float*,float*);
extern "C" void set(int,float,float*);

int main(int argc, char **argv)
{
  float *x, *y, tmp;
  int n = 1<<20;

  hipMalloc((void**)&x,(size_t)n*sizeof(float));
  hipMalloc((void**)&y,(size_t)n*sizeof(float));

  set(n,1.0f,x);
  set(n,0.0f,y);

  saxpy(n, 2.0, x, y);
  hipMemcpy(&tmp,y,(size_t)sizeof(float),hipMemcpyDeviceToHost);
  printf("%f\n",tmp);
  return 0;
}
